#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

extern "C" {
#include "utilities.h"
#include "graphLib.h"
#include "settings.h"
}

// cuda enforces an upper limit of 512 threads per blocks 
#define THREADS_PER_BLOCK 500

// cuda enforces an upper limit of 65xxx blocks per each grid dimension
#define BLOCKS_PER_DIM 65000

#define TOTAL_LIVE_OBJECTS 5


__device__ int getThreadId() {
  return 
    threadIdx.x +
    blockIdx.x * THREADS_PER_BLOCK +
    blockIdx.y * BLOCKS_PER_DIM * THREADS_PER_BLOCK +
    blockIdx.z * BLOCKS_PER_DIM * BLOCKS_PER_DIM * THREADS_PER_BLOCK;
}

__device__ int totalMarked(int *visited) {
	int marked = 0, i;
	for (i=0; i<MAX_NODES; i++)
		if (visited[i])
			marked++;

	return marked;
}

__global__ void gcKernel(int numNodes, int *graph, int *visited) {
	int index = getThreadId();
	if (index > numNodes)
		return;

	int *node = graph + (index*MAX_CHILDREN);
	int numMarked=0, i;
	while (numMarked < TOTAL_LIVE_OBJECTS) { // hardcoded for now
		if (visited[index]) { // I've been visited
			i = 0;
			while (node[i]) // notify all my children
				visited[node[i++]] = 1;
			return; // I'm done now
		}
		numMarked = totalMarked(visited);
	}
	return;
}


void computeGridDimesions(int totalThreads, dim3 *dimensions);

int main() {
	double startTime, stopTime;
	int i, *graph_d, *visited_d;
	dim3 gridDimensions;
	int graphSize = sizeof(int)*MAX_NODES*MAX_CHILDREN, visitedSize = sizeof(int)*MAX_NODES;

	initialize();
	readInput();

	// initialize visited array with rootset
	i=0;
	while (rootset[i])
		visited[rootset[i++]] = 1;

	// allocate and load graph and visited arrays on gpu
	assert(hipMalloc(&graph_d, graphSize) == hipSuccess);
	assert(hipMalloc(&visited_d, visitedSize) == hipSuccess);
	assert(hipMemcpy(graph_d, nodes, graphSize, hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy(visited_d, visited, visitedSize, hipMemcpyHostToDevice) == hipSuccess);
	
	computeGridDimesions(numNodes, &gridDimensions);

	// collect garbage
	currentTimestamp(&startTime);
	gcKernel<<<gridDimensions, THREADS_PER_BLOCK>>>(numNodes, graph_d, visited_d);
	currentTimestamp(&stopTime);

	// copy visited back
	assert(hipMemcpy(visited, visited_d, visitedSize, hipMemcpyDeviceToHost) == hipSuccess);

	// free graph and visited arrays on gpu
	hipFree(graph_d); 
	hipFree(visited_d);

	printAlive();
	//printf("%lf\n", stopTime-startTime);
	return 0;
}

void computeGridDimesions(int totalThreads, dim3 *dimensions) {
  int remainingBlocks = ceil((double)totalThreads / (double)THREADS_PER_BLOCK);

  dimensions->x = min(remainingBlocks, BLOCKS_PER_DIM);
  remainingBlocks = ceil((double)remainingBlocks / BLOCKS_PER_DIM);

  dimensions->y = min(remainingBlocks, BLOCKS_PER_DIM);
  remainingBlocks = ceil((double)remainingBlocks / BLOCKS_PER_DIM);

  dimensions->z = min(remainingBlocks, BLOCKS_PER_DIM);
  remainingBlocks = ceil((double)remainingBlocks / BLOCKS_PER_DIM);

  assert(remainingBlocks == 1); // otherwise, our problem is too big for grid!
}
